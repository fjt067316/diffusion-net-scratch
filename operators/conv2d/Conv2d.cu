#include "hip/hip_runtime.h"
#pragma once

#include <cstdio>
#include <hip/hip_runtime.h>
#include <cassert>
#include <initializer_list>
#include <utility>
#include <cmath>

#include "Conv2d.h"

#define CUDA_CHECK(call) \
    do { \
        hipError_t error = call; \
        if (error != hipSuccess) { \
            fprintf(stderr, "CUDA error at %s:%d: %s\n", \
                    __FILE__, __LINE__, hipGetErrorString(error)); \
            exit(EXIT_FAILURE); \
        } \
    } while (0)


__host__ __device__ float getElement(float *arr, int i) {
    return arr[i];
}

__host__ __device__ float getElement(float *arr, int* dims, int i, int j) {
    return arr[i * dims[1] + j];
}

__host__ __device__ float getElement(float *arr, int* dims, int i, int j, int k) {
    return arr[i * dims[1] * dims[2] + j * dims[2] + k];
}

__host__ __device__ float getElement(float *arr, int* dims, int i, int j, int k, int l) {
    return arr[i * dims[1] * dims[2] * dims[3] + j * dims[2] * dims[3] + k * dims[3] + l];
}

__host__ __device__ float getIdx(int* dims, int i, int j, int k, int l) {
    return i * dims[1] * dims[2] * dims[3] + j * dims[2] * dims[3] + k * dims[3] + l;
}


__global__ void conv_forward(float* input, float* output, float* weights, float* bias, int* in_dims, int* out_dims, int* w_dims, int padding, int stride, bool use_bias, bool use_relu) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int c_out = blockIdx.z;
    
    int batch_size = in_dims[0],
       input_channels = in_dims[1],
       output_channels = out_dims[1],
       height = in_dims[2],
       width = in_dims[3],
       filter_size = w_dims[2];

    if (x >= width+padding || y >= height+padding || x+filter_size-1 >= width+padding || y+filter_size-1 >= height+padding || c_out >= output_channels || (x%stride) != 0 || (y%stride) != 0){
        return;
    }

    float bias_val = use_bias ? bias[c_out] : 0;

    for(int b=0; b < batch_size; b++){
        float sum = bias_val;
        for(int c=0; c<input_channels; c++){
            for(int row=y; row<y+filter_size; row++){
                for(int col=x; col<x+filter_size; col++){
                    if(row < padding || col < padding || row >= height+padding || col >= width+padding ){ // padding guard
                        continue;
                    } else{
                        sum += getElement(input, in_dims, b, c, row - padding, col - padding) * getElement(weights, w_dims, c_out, c, row - y, col - x);
                    }
                }
            }
        }

        int idx = getIdx(out_dims, b, c_out, y/stride, x/stride);
        if(use_relu && sum < 0){
            output[idx] = 0;
        }else{
            output[idx] = sum;
        }

    }
}


__global__ void conv_dldz_next(float* aug_w, float* dldz_next, float* dldz, int* w_dims, int* next_dims, int* dz_dims, int padding, int stride, int output_channels) {
    // one thread per dl_dz_next element (excluding batch dim)
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int in_ch = blockIdx.z % output_channels;
    int batch = blockIdx.z / output_channels; // should be able to do this 12288 < 65000
    
    // we want a 2d filter to sweep across all 3 channels and compute the 2d*2d conv on each layer
    // ie 3d input 2d weight (one filter / layer of dldz) 3d output
    int batch_size = next_dims[0];
    int filter_size_h = dz_dims[2];
    int filter_size_w = dz_dims[3];
    int n_filter = w_dims[0];
    int height = aug_w[2];
    int width = aug_w[3];

    if (x+filter_size-1 >= width || y+filter_size-1 >= height || in_ch >= next_dims[1] || (x%stride) != 0 || (y%stride) != 0){
        return;
    }
    // we will just perform a 2d by 2d convoultion and spawn 1 thread for every element in output and across every filter (excluding batch size)
    // for(int b=0; b<batch_size; b++){
    float sum = 0;
    for(int filter_idx=0; filter_idx<n_filter; filter++){
        for(int row=y; row<y+filter_size_h; row++){
            for(int col=x; col<x+filter_size_w; col++){
                sum += getElement(dldz, dz_dims, batch, filter_idx, row-y, col-x) * getElement(aug_w, w_dims, filter_idx, in_ch, row, col);
            }
        }
    }

    int idx = getIdx(next_dims, batch, in_ch, y/stride, x/stride);
    dldz_next[idx] = sum;
    // }
    
}

/*
assumes that data is already on gpu
*/
Tensor<float, 4> Conv2d::forward(Tensor<float,4> &input){
    // temp
    this->input = input;

    assert(input.dim(1) == this->input_channels);
    
    int batch_size = input.dim(0), height = input.dim(2), width = input.dim(3);

    int out_width = (width - filter_size + 2 * padding) / stride + 1;
    int out_height = (height - filter_size + 2 * padding) / stride + 1;

    Tensor<float, 4> output({batch_size, output_channels, out_height, out_width}, false); // do_allocs=false
    float* d_out;
    hipMalloc(&d_out, output.size * sizeof(float));
    output.data = d_out;
    // int N = 768;// out_height*out_width;
    int tds = 16; // 2d block -> 256 threads per thread block
    int block_height = (int) ceil((height + 2*padding) / tds);
    int block_width = (int) ceil((width + 2*padding) / tds);

    dim3 threadDim(tds, tds, 1);
    dim3 blockDim(block_width, block_height, output_channels);

    conv_forward <<<blockDim, threadDim>>>(input.data, output.data, weights.data, bias.data, input.d_dims, output.d_dims, weights.d_dims, padding, stride, this->use_bias, this->use_relu);

    Tensor<float, 4> result(batch_size, output_channels, out_height, out_width);

    hipMemcpy(result.data, output.data, sizeof(float) * output.size, hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    // Free the device memory allocated for input tensor
    hipFree(d_in);

    return result;
}


// Backprop stuff below

// this is correct stop looking at it
__global__ void get_dw(float* input, float* dLdZ, float* output, int* in_dim, int* dz_dim, int* out_dim, int padding, int stride, int n_filters){
    int x = blockIdx.z * blockDim.z + threadIdx.z; 
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int z_idx = blockIdx.x*blockDim.x + threadIdx.x; // we reserve x idx which can hold a lot of blocks for our longest dim
    int filter_idx = z_idx % n_filters;
    int channel = z_idx / n_filters;


    int filter_size = out_dim[2];
    int batch_size = in_dim[0];

    if(x >= filter_size || y >= filter_size || filter_idx >= dz_dim[1] || channel >= in_dim[1]){
        return;
    }

    int in_w = in_dim[3] + 2*padding;
    int in_h = in_dim[2] + 2*padding;

    int w_moves = (in_w-filter_size)/stride+1;
    int h_moves = (in_h-filter_size)/stride+1;

    
    assert(h_moves == dz_dim[2]);
    assert(w_moves == dz_dim[3]);
    float dw = 0;


    for(int b=0; b<batch_size; b++){
        int dz_idx = getIdx(dz_dim, b, filter_idx, 0,0); // should be 1 dz for every move made 

        for(int i=0, y_off=0; i<h_moves; i++, y_off += stride){ 
            for(int j=0, x_off=0; j<w_moves; j++, x_off += stride, dz_idx++){

            if(idx_x < padding || idx_y < padding || idx_x >= in_w-padding || idx_y >= in_h-padding){
                continue;
            }
            
            int idx_x = x+x_off;
            int idx_y = y+y_off;

            float in = getElement(input, in_dim, b, channel, idx_y-padding, idx_x-padding);
            dw += in; // maybe make it dw += in; then at end dw *= dz
            }
        }
    }
    dw *= * dZ;
    int idx = getIdx(out_dim, filter_idx, channel, y, x);
    output[idx] = dw / batch_size;

}

__global__ void rotate180(float* input, float* output, int* dims) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int filter_idx = blockIdx.z;


    int height = dims[2];
    int width = dims[3];
    int num_channels = dims[1];

    if (x < width && y < height) {
        int in_index = filter_idx * width * height * num_channels + y * width + x;
        int out_index = filter_idx * width * height * num_channels + (height - y - 1) * width + (width - x - 1);
        output[out_index] = input[in_index];
    }
}



__global__ void pad_image_tranpose(float* input, float* output, int* in_dims, int* out_dims, int z, int pp){
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    // int c_out = blockIdx.z * blockDim.z;
    int b = blockIdx.z;

    int batch_size = in_dims[0],
        input_channels = in_dims[1],
        output_channels = out_dims[1],
        out_h = out_dims[2],
        out_w = out_dims[3];

    assert(input_channels == output_channels);

    if(x >= out_w || y >= out_h || b >= batch_size){
        return;
    }

    // check if output index should have an input value copied or be filled with zeros
    int x_valid = (x-pp) % (z+1);
    int y_valid = (y-pp) % (z+1);

    int idx = getIdx(out_dims, b, 0, y, x);
    int off = out_h*out_w;
    // zeros condition
    if(x < pp || y < pp || x >= out_w-pp || y >= out_h-pp || x_valid != 0 || y_valid != 0){
        for(int i=0; i<input_channels; i++){
            output[idx+i*off] = 0;
        }
        return;
    }

    // else fill with input value 
    int in_x = (x-pp) / (z+1);
    int in_y = (y-pp) / (z+1);
    // printf("row %d col %d val %d", in_y, in_x, getElement(input, in_dims, b, c_out, in_y, in_x));
    for(int i=0; i<input_channels; i++){
        // printf("added %f from %d %d to %d %d\n",getElement(input, in_dims, b, i, in_y, in_x), in_y, in_x, y, x );
        output[idx + i*off] = getElement(input, in_dims, b, i, in_y, in_x); 

    }
}

Tensor<float, 4> conv_transpose_2d_dldz(Tensor<float,4> &input, Tensor<float, 4> weights, Tensor<float, 1> bias, int padding, int stride, bool rotate180_weights , bool use_bias = false){

    if(rotate180_weights){
        float* rot;
        hipMalloc(&rot, weights.size*sizeof(float));

        int rotTds = 16; // 2d block -> 256 threads per thread block
        int block_height = (int)ceil((double)weights.dim(2) / (double)rotTds);
        int block_width = (int)ceil((double)weights.dim(3) / (double)rotTds);

        dim3 rotBlockDim(rotTds, rotTds); // You may adjust block dimensions according to your matrix size
        dim3 rotGridDim(block_width, block_height, weights.dim(0));
        rotate180<<<rotGridDim, rotBlockDim>>>(weights.data, rot, weights.d_dims);
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());
        weights.data = rot;
    }

    int batch_size = input.dim(0), in_channels = input.dim(1), height = input.dim(2), width = input.dim(3), filter_size = weights.dim(2);
    int output_channels = weights.dim(0);
    // insert z zeros between input vals and pp zeros around edge
    int z = stride-1;
    int pp = filter_size-padding-1;

    // create temp input to perform conv on because I dont want to do weird atomic add transpose thing
    int h_tmp = 2*pp+z*(height-1) + height;
    int w_tmp = 2*pp+z*(width-1) + width;

    // create and pad tmp array
    Tensor<float, 4> tmp({batch_size, in_channels, h_tmp, w_tmp}, true, true);

    int tds = 16; // 2d block -> 256 threads per thread block
    int block_height = (int)ceil(((double)h_tmp) / (double)tds);
    int block_width = (int)ceil(((double)w_tmp) / (double)tds);

    dim3 threadDim(tds, tds, 1);
    dim3 blockDim(block_width, block_height, batch_size);

    CUDA_CHECK(hipGetLastError()); // Ensure there's no previous kernel launch errors
    CUDA_CHECK(hipDeviceSynchronize());

    pad_image_tranpose<<<blockDim, threadDim>>>(input.data, tmp.data, input.d_dims, tmp.d_dims, z, pp);

    CUDA_CHECK(hipGetLastError()); // Ensure there's no previous kernel launch errors
    CUDA_CHECK(hipDeviceSynchronize());
    // create output array for convolution
    int out_height = (height-1) * stride + filter_size-2*padding;
    int out_width = (width-1) * stride + filter_size-2*padding;

    Tensor<float, 4> output({batch_size, output_channels, out_height, out_width}, false); // output for actual convolution
    hipMalloc(&output.data, output.size*sizeof(float));

    tds = 16; // 2d block -> 256 threads per thread block
    block_height = (int) ceil((double)out_height / (double)tds);
    block_width = (int) ceil((double)out_width / (double)tds);
    block_depth = (int) ceil((double)out_width / (double)tds);

        // one thread per dl_dz_next element (excluding batch dim)
    // int x = blockIdx.x * blockDim.x + threadIdx.x;
    // int y = blockIdx.y * blockDim.y + threadIdx.y;
    // int in_ch = threadIdx.z;
    // int batch = blockIdx.z; // should be able to do this
    // float* aug_w, float* dldz_next, float* dldz, int* w_dims, int* next_dims, int* dz_dims, int padding, int strid
    dim3 threadDimOut(tds, tds, 1);
    dim3 blockDimOut(block_width, block_height, output_channels*batch_size); // output_channel = num weights

    CUDA_CHECK(hipGetLastError()); 
    CUDA_CHECK(hipDeviceSynchronize());
    conv_dldz_next<<<blockDimOut, threadDimOut>>>(tmp.data, output.data, weights.data, weights.d_dims, output.d_dims, tmp.d_dims, 0, 1, output_channels);
    // conv_forward <<<blockDimOut, threadDimOut>>>(tmp.data, output.data, weights.data, bias.data, tmp.d_dims, output.d_dims, weights.d_dims,0, 1, use_bias); // no padding argument
    CUDA_CHECK(hipGetLastError()); // Ensure there's no previous kernel launch errors
    CUDA_CHECK(hipDeviceSynchronize());
    hipFree(tmp.data);

    return output;
}

// __global__ void remove_pad(float* padded_in, float* out, int* in_dim, int* out_dim, int padding){
//     int x = blockIdx.x * blockDim.x + threadIdx.x;
//     int y = blockIdx.y * blockDim.y + threadIdx.y;
//     int in_ch = theadIdx.z;
//     int batch = blockIdx.z;

//     // we want to 

//     if(x >= in_dim[3]-padding || y >= in_dim[2]-padding || batch >= in_dim[0] || in_ch >= in_dim[1] || x < padding || y < padding){
//         return;
//     }

//     int unpad_x = x - padding;
//     int unpad_y = y-padding;

//     float* val = getElement(padded_in, in_dim, batch, in_ch, y, x);

//     int idx = getIdx(in_dim, batch, in_ch, unpad_y, unpad_x);

//     out[idx] = val;

// }


__global__ void apply_dw(float* weights, float* dw, int* w_dims, int n_filters){
    int x = blockIdx.z * blockDim.z + threadIdx.z; 
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int z_idx = blockIdx.x*blockDim.x + threadIdx.x; // we reserve x idx which can hold a lot of blocks for our longest dim
    int filter_idx = z_idx % n_filters;
    int channel = z_idx / n_filters;

    int idx = getIdx(w_dims, filter_n, filter_ch, y, x);

    weights[idx] -= 0.0001*dw[idx];
}

__global__ void apply_relu(float* dLdZ, float* input, int* in_dims){
    // 4d dLdZ because conv
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int channel = theadIdx.z;
    int batch = blockIdx.z;

    int idx = getIdx(in_dims, batch, channel, y, x);
    float val = input[idx];

    if(val < 0){
        dLdZ[idx] = 0;
    }else{
        dLdZ[idx] = val;
    }
}

Tensor<float, 4> Conv2d::backward(Tensor<float,4> &dLdZ){

    if(this->use_relu){ // apply relu backwards
        int tds = 16;
        int block_height = (int)ceil(((double)input.dim(2)) / (double)tds);
        int block_width = (int)ceil(((double)input.dim(3)) / (double)tds);
        int batch_size = input.dim(0);
    
        dim3 threadDimRelu(tds, tds, input_channels);
        dim3 blockDimRelu(block_width, block_height, batch_size);
        apply_relu<<<blockDimRelu, threadDimRelu>>>(dLdZ.data, input.data, input.d_dims);
    }

    Tensor<float, 4> dLdZ_next = conv_transpose_2d_dldz(this->weights, dLdZ, this->padding, this->stride, true);
    Tensor<float, 4> dWdZ({output_channels, input_channels, filter_size, filter_size}, true, true);

    tds = 8; // 2d block -> 256 threads per thread block
    block_height = (int)ceil(((double)filter_size) / (double)tds);
    block_width = (int)ceil(((double)filter_size) / (double)tds);

    dim3 threadDim(3, tds, tds);
    dim3 blockDim(input_channels*output_channels, block_height, block_width);
    CUDA_CHECK(hipGetLastError()); 
    CUDA_CHECK(hipDeviceSynchronize());
    // sum and average weights across mini-batch before updating weights
    get_dw<<<blockDim, threadDim>>>(this->input.data, dLdZ.data, dWdZ.data, input.d_dims, dLdZ.d_dims, dWdZ.d_dims, padding, stride, output_channels);
    CUDA_CHECK(hipGetLastError()); 
    CUDA_CHECK(hipDeviceSynchronize());
    // cut out padding before returning to next layer
    assert(input.dim(0) == dLdZ.dim(0));
    assert(input.dim(1) == dLdZ.dim(1));
    assert(input.dim(2) == dLdZ_next.dim(2));
    assert(input.dim(3) == dLdZ_next.dim(3));


    // int batch_size = dLdZ.dim(0);
    // block_height = (int)ceil(((double)dLdZ.dim(2)) / (double)tds);  // one thread per padded input height width row 
    // block_width = (int)ceil(((double)dLdZ.dim(3)) / (double)tds);

    // dim3 padThreadDim(tds, tds, input_channels);
    // dim3 padBlockDim(block_width, block_height, batch_size);

    // Tensor<float, 4> unpad_dldz({input.dim(0), input.dim(1), input.dim(2), input.dim(3)}, true, true);
    // CUDA_CHECK(hipGetLastError()); 
    // CUDA_CHECK(hipDeviceSynchronize());
    // remove_pad<<padBlockDim, padThreadDim>>(dLdZ.data, unpad_dldz.data, dLdZ.d_dims, unpad_dldz.d_dims, padding);
    // CUDA_CHECK(hipGetLastError()); 
    // CUDA_CHECK(hipDeviceSynchronize());

    // dWdZ.toHost();
    // input.toHost();
    // for(int i=0; i<4; i++){
    //     printf(" %f ", dWdZ.data[i]);
    // }
    // printf("\n");
    // pass back 4D tensor with individual grads pre input

    tds = 8; // small thread count as filters are small but filter_n is big
    block_height = (int)ceil(((double)filter_size) / (double)tds);
    block_width = (int)ceil(((double)filter_size) / (double)tds);

    dim3 threadDimDw(3, tds, tds);
    dim3 blockDimDw( input_channels*output_channels, block_height, block_width); // move biggest dims onto block x as it can hold a fuck tonne of blocks

    apply_dw<<<blockDimDw, threadDimDw>>>(this->weights.data, dWdZ.data, this->weights.d_dims, output_channels);
    CUDA_CHECK(hipGetLastError()); 
    CUDA_CHECK(hipDeviceSynchronize());

    return dLdZ_next;

}